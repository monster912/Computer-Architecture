/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
        {
            double sum = 0;

            for (unsigned int k = 0; k < wA; ++k)
            {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }

            C[i * wB + j] = (float)sum;
        }
}

__global__ void
matrixMul_naive( float* C, float* A, float* B, int wA, int wB)
{
int Row = blockIdx.y*blockDim.y+threadIdx.y;
int Col = blockIdx.x*blockDim.x+threadIdx.x;

if ((Row < wA) && (Col<wB))
  {
	float Cvalue=0;
	for (int k = 0 ; k < wA; ++k) Cvalue += A[Row*wA+k]*B[k*wB+Col];
	C[Row*wB+Col]=Cvalue; 
 }

}





void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int size_C = dimsC.x * dimsC.y;
    unsigned int mem_size_C = size_C * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    matrixMulCPU(reference, h_A, h_B, dimsA.y, dimsA.x, dimsB.x);
    printf("done.\n");


    hipMalloc((void **) &d_A, mem_size_A);
    hipMalloc((void **) &d_B, mem_size_B);
    hipMalloc((void **) &d_C, mem_size_C);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

	// Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    matrixMul_naive<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    hipEventCreate(&start);

    hipEvent_t stop;
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
		matrixMul_naive<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    // Record the stop event
    hipEventRecord(stop, NULL);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);


    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);


    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
		hipFree(d_C);

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    // Use a larger block size for Fermi and above
    int block_size = 32;

    dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
    dim3 dimsB(5*4*block_size, 5*2*block_size, 1);

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
